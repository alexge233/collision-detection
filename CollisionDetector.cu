#include "hip/hip_runtime.h"
#include "include.cuh"

__constant__ uint numAABB_const;
__constant__ realV bin_size_vec_const;
__constant__ uint last_active_bin_const;
__constant__ uint number_of_contacts_possible_const;

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// Constructor, must pass the aabb_data (puts user in control)
CollisionDetector::CollisionDetector(custom_vector<realV> aabb_data) {
	number_of_contacts_possible = 0;
	val = 0;
	last_active_bin = 0;
	number_of_bin_intersections = 0;
	this->aabb_data = aabb_data;

	numAABB = aabb_data.size()/2; // TODO: Should make aabb_data organization less confusing, compiler should switch depending on if the user passes a host/device vector
	bins_per_axis = F3(100, 100, 100); // TODO: Should be able to tune this, it's nice to have as a parameter though!
	// TODO: As the collision detection is progressing, we should free up vectors that are no longer being used! For example, Bin_Intersections is only used in steps 4&5
}

int CollisionDetector::updateBoundingBoxes(custom_vector<realV> aabb_data) {
	this->aabb_data = aabb_data;
	numAABB = aabb_data.size()/2;
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
template<class T>
inline int3 __host__ __device__ HashMax( //CHANGED: For maximum point, need to check if point lies on edge of bin (TODO: Hmm, fmod still doesn't work completely)
        const T &A,
        const realV & bin_size_vec) {
    int3 temp;
    temp.x = A.x / bin_size_vec.x;
    if(!fmod(A.x,bin_size_vec.x)) temp.x--;
    temp.y = A.y / bin_size_vec.y;
    if(!fmod(A.y,bin_size_vec.y)) temp.y--;
    temp.z = A.z / bin_size_vec.z;
    if(!fmod(A.z,bin_size_vec.z)) temp.z--;

    //cout << temp.x << " " << temp.y << " " << temp.z << endl;
    return temp;
}

template<class T>
inline int3 __host__ __device__ HashMin(
        const T &A,
        const realV & bin_size_vec) {
    int3 temp;
    temp.x = A.x / bin_size_vec.x;
    temp.y = A.y / bin_size_vec.y;
    temp.z = A.z / bin_size_vec.z;

    //cout << temp.x << " " << temp.y << " " << temp.z << endl;
    return temp;
}

template<class T>
inline uint __host__ __device__ Hash_Index(
        const T &A) {
    return ((A.x * 73856093) ^ (A.y * 19349663) ^ (A.z * 83492791));
}

//Function to Count AABB Bin intersections
inline void __host__ __device__ function_Count_AABB_BIN_Intersection(
        const uint & index,
        const realV* aabb_data,
        const realV & bin_size_vec,
        const uint & number_of_particles,
        uint* Bins_Intersected) {
    int3 gmin = HashMin(aabb_data[index], bin_size_vec);
    int3 gmax = HashMax(aabb_data[index + number_of_particles], bin_size_vec);
    Bins_Intersected[index] = (gmax.x - gmin.x + 1) * (gmax.y - gmin.y + 1) * (gmax.z - gmin.z + 1);
}
//--------------------------------------------------------------------------
__global__ void device_Count_AABB_BIN_Intersection(
        const float3* aabb_data,
        uint* Bins_Intersected) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, numAABB_const)
    function_Count_AABB_BIN_Intersection(index, aabb_data, bin_size_vec_const, numAABB_const, Bins_Intersected);

}
//--------------------------------------------------------------------------
void CollisionDetector::host_Count_AABB_BIN_Intersection(
        const realV* aabb_data,
        uint* Bins_Intersected) {
#pragma omp parallel for schedule(guided)
    for (int i = 0; i < numAABB; i++) {
        function_Count_AABB_BIN_Intersection(i, aabb_data, bin_size_vec, numAABB, Bins_Intersected);
    }
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//Function to Store AABB Bin Intersections

inline void __host__ __device__ function_Store_AABB_BIN_Intersection(
        const uint & index,
        const realV* aabb_data,
        const uint* Bins_Intersected,
        const realV & bin_size_vec,
        const uint & number_of_particles,
        uint * bin_number,
        uint * body_number) {
    uint count = 0, i, j, k;
    int3 gmin = HashMin(aabb_data[index], bin_size_vec);
    int3 gmax = HashMax(aabb_data[index + number_of_particles], bin_size_vec);
    uint mInd = (index == 0) ? 0 : Bins_Intersected[index - 1];

    for (i = gmin.x; i <= gmax.x; i++) {
        for (j = gmin.y; j <= gmax.y; j++) {
            for (k = gmin.z; k <= gmax.z; k++) {
                bin_number[mInd + count] = Hash_Index(U3(i, j, k));
                body_number[mInd + count] = index;
                count++;
            }
        }
    }
}
//--------------------------------------------------------------------------
__global__ void device_Store_AABB_BIN_Intersection(
        const float3* aabb_data,
        const uint* Bins_Intersected,
        uint * bin_number,
        uint * body_number) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, numAABB_const);
    function_Store_AABB_BIN_Intersection(index, aabb_data, Bins_Intersected, bin_size_vec_const, numAABB_const, bin_number, body_number);
}
//--------------------------------------------------------------------------

void CollisionDetector::host_Store_AABB_BIN_Intersection(
        const realV* aabb_data,
        const uint* Bins_Intersected,
        uint * bin_number,
        uint * body_number) {
#pragma omp parallel for schedule(guided)
    for (int i = 0; i < numAABB; i++) {
        function_Store_AABB_BIN_Intersection(i, aabb_data, Bins_Intersected, bin_size_vec, numAABB, bin_number, body_number);
    }
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//Function to count AABB AABB intersection

inline void __host__ __device__ function_Count_AABB_AABB_Intersection(
        const uint & index,
        const realV* aabb_data,
        const uint & number_of_particles,
        const uint * bin_number,
        const uint * body_number,
        const uint * bin_start_index,
        uint* Num_ContactD) {
    uint end = bin_start_index[index], count = 0, i = (!index) ? 0 : bin_start_index[index - 1];
    uint tempa, tempb;
    AABB A, B;
    for (; i < end; i++) {
        tempa = body_number[i];
        A.min = aabb_data[tempa];
        A.max = aabb_data[tempa + number_of_particles];
        for (int k = i + 1; k < end; k++) {
            tempb = body_number[k];
            B.min = aabb_data[tempb];
            B.max = aabb_data[tempb + number_of_particles];
            bool inContact = (A.min.x <= B.max.x && B.min.x <= A.max.x) && (A.min.y <= B.max.y && B.min.y <= A.max.y) && (A.min.z <= B.max.z && B.min.z <= A.max.z);
            if (inContact) count++;
        }
    }
    Num_ContactD[index] = count;
}
//--------------------------------------------------------------------------
__global__ void device_Count_AABB_AABB_Intersection(
        const realV* aabb_data,
        const uint * bin_number,
        const uint * body_number,
        const uint * bin_start_index,
        uint* Num_ContactD) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, last_active_bin_const);
    function_Count_AABB_AABB_Intersection(index, aabb_data, numAABB_const, bin_number, body_number, bin_start_index, Num_ContactD);
}

//--------------------------------------------------------------------------
void CollisionDetector::host_Count_AABB_AABB_Intersection(
        const realV* aabb_data,
        const uint * bin_number,
        const uint * body_number,
        const uint * bin_start_index,
        uint* Num_ContactD) {
#pragma omp parallel for schedule(guided)
    for (int i = 0; i < last_active_bin; i++) {
        function_Count_AABB_AABB_Intersection(i, aabb_data, numAABB, bin_number, body_number, bin_start_index, Num_ContactD);
    }
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
//Function to store AABB-AABB intersections
inline void __host__ __device__ function_Store_AABB_AABB_Intersection(
        const uint & index,
        const realV* aabb_data,
        const uint & number_of_particles,
        const uint * bin_number,
        const uint * body_number,
        const uint * bin_start_index,
        const uint* Num_ContactD,
        long long* potential_contacts) {
    uint end = bin_start_index[index], count = 0, i = (!index) ? 0 : bin_start_index[index - 1], Bin = bin_number[index];
    uint offset = (!index) ? 0 : Num_ContactD[index - 1];
    if (end - i == 1) {
        return;
    }
    uint tempa, tempb;
    AABB A, B;
    for (; i < end; i++) {
        ;
        tempa = body_number[i];
        A.min = aabb_data[tempa];
        A.max = aabb_data[tempa + number_of_particles];

        for (int k = i + 1; k < end; k++) {
            tempb = body_number[k];

            B.min = aabb_data[tempb];
            B.max = aabb_data[tempb + number_of_particles];

            bool inContact = (A.min.x <= B.max.x && B.min.x <= A.max.x) && (A.min.y <= B.max.y && B.min.y <= A.max.y) && (A.min.z <= B.max.z && B.min.z <= A.max.z);
            if (inContact == true) {

                int a = tempa;
                int b = tempb;
                if (b < a) {
                    int t = a;
                    a = b;
                    b = t;
                }
                potential_contacts[offset + count] = ((long long) a << 32 | (long long) b); //the two indicies of the objects that make up the contact
                count++;
            }
        }
    }
}
//--------------------------------------------------------------------------
__global__ void device_Store_AABB_AABB_Intersection(
        const float3* aabb_data,
        const uint * bin_number,
        const uint * body_number,
        const uint * bin_start_index,
        const uint* Num_ContactD,
        long long* potential_contacts) {
    INIT_CHECK_THREAD_BOUNDED(INDEX1D, last_active_bin_const);

    function_Store_AABB_AABB_Intersection(index, aabb_data, numAABB_const, bin_number, body_number, bin_start_index, Num_ContactD, potential_contacts);
//--------------------------------------------------------------------------
}
void CollisionDetector::host_Store_AABB_AABB_Intersection(
        const float3* aabb_data,
        const uint * bin_number,
        const uint * body_number,
        const uint * bin_start_index,
        const uint* Num_ContactD,
        long long* potential_contacts) {
#pragma omp parallel for schedule(guided)
    for (int index = 0; index < last_active_bin; index++) {
        function_Store_AABB_AABB_Intersection(index, aabb_data, numAABB, bin_number, body_number, bin_start_index, Num_ContactD, potential_contacts);
    }
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
// use spatial subdivision to detect the list of POSSIBLE collisions (let user define their own narrow-phase collision detection)
int CollisionDetector::detectPossibleCollisions() {
	double startTime = omp_get_wtime();

	// STEP 1: Initialization TODO: this could be put in the constructor
#ifdef SIM_ENABLE_GPU_MODE
	// set the default cache configuration on the device to prefer a larger L1 cache and smaller shared memory
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Count_AABB_BIN_Intersection), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Store_AABB_BIN_Intersection), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Count_AABB_AABB_Intersection), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(device_Store_AABB_AABB_Intersection), hipFuncCachePreferL1);
	COPY_TO_CONST_MEM(numAABB);
#endif
	potentialCollisions.clear();
	// END STEP 1

	// STEP 2: determine the bounds on the total space and subdivide based on the bins per axis
	bbox init = bbox(aabb_data[0], aabb_data[0]); // create a zero volume bounding box using the first set of aabb_data (??)
	bbox_transformation unary_op;
	bbox_reduction binary_op;
	bbox result = thrust::transform_reduce(aabb_data.begin(), aabb_data.end(), unary_op, init, binary_op);
	min_bounding_point = result.first;
	max_bounding_point = result.second;
	global_origin = fabs(min_bounding_point); // TODO: Look closely at this to see if correct
	bin_size_vec = (fabs(max_bounding_point + fabs(min_bounding_point)));
	bin_size_vec = bin_size_vec/bins_per_axis; //CHANGED: this was supposed to be reversed
	thrust::transform(aabb_data.begin(), aabb_data.end(), thrust::constant_iterator<realV>(global_origin), aabb_data.begin(), thrust::minus<realV>()); //CHANGED: Should be a minus
	cout << "Global Origin: (" << global_origin.x << ", " << global_origin.y << ", " << global_origin.z << ")"<< endl;
	cout << "Maximum bounding point: (" << max_bounding_point.x << ", " << max_bounding_point.y << ", " << max_bounding_point.z << ")"<< endl;
	cout << "Bin size vector: (" << bin_size_vec.x << ", " << bin_size_vec.y << ", " << bin_size_vec.z << ")"<< endl;
	// END STEP 2

	// STEP 3: Count the number AABB's that lie in each bin, allocate space for each AABB
	Bins_Intersected.resize(numAABB); // TODO: how do you know how large to make this vector?
	// TODO: I think there is something wrong with the hash function...
#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(bin_size_vec);
	device_Count_AABB_BIN_Intersection __KERNEL__(BLOCKS(numAABB),THREADS)(CASTF3(aabb_data), CASTU1( Bins_Intersected));
#else
	host_Count_AABB_BIN_Intersection(aabb_data.data(), Bins_Intersected.data());
#endif
	Thrust_Inclusive_Scan_Sum(Bins_Intersected, number_of_bin_intersections);
	cout << "Number of bin intersections: " << number_of_bin_intersections << endl;
	bin_number.resize(number_of_bin_intersections);
	body_number.resize(number_of_bin_intersections);
	bin_start_index.resize(number_of_bin_intersections);
	// END STEP 3

	// STEP 4: Indicate what bin each AABB belongs to, then sort based on bin number
#ifdef SIM_ENABLE_GPU_MODE
	device_Store_AABB_BIN_Intersection __KERNEL__(BLOCKS(numAABB),THREADS)(CASTF3(aabb_data), CASTU1( Bins_Intersected), CASTU1( bin_number), CASTU1( body_number));
#else
	host_Store_AABB_BIN_Intersection(aabb_data.data(), Bins_Intersected.data(),
			bin_number.data(), body_number.data());
#endif
	Thrust_Sort_By_Key(bin_number, body_number);
	Thrust_Reduce_By_KeyA(last_active_bin, bin_number, bin_start_index);

		//QUESTION: I have no idea what is going on here
	val =
			bin_start_index[thrust::max_element(bin_start_index.begin(),
					bin_start_index.begin() + last_active_bin)
					- bin_start_index.begin()];
	if (val > 50) {
		bins_per_axis = bins_per_axis * 1.1;
	} else if (val < 25 && val > 1) {
		bins_per_axis = bins_per_axis * .9;
	}
	bin_start_index.resize(last_active_bin);
	cout << "Last active bin: " << last_active_bin << endl;
	Thrust_Inclusive_Scan(bin_start_index);
	Num_ContactD.resize(last_active_bin);
	// END STEP 4

	// STEP 5: Count the number of AABB collisions
#ifdef SIM_ENABLE_GPU_MODE
	COPY_TO_CONST_MEM(last_active_bin);
	device_Count_AABB_AABB_Intersection __KERNEL__(BLOCKS(last_active_bin),THREADS)(
			CASTF3(aabb_data),
			CASTU1(bin_number),
			CASTU1(body_number),
			CASTU1(bin_start_index),
			CASTU1(Num_ContactD));
#else
	host_Count_AABB_AABB_Intersection(aabb_data.data(), bin_number.data(), body_number.data(), bin_start_index.data(), Num_ContactD.data());
#endif
	Thrust_Inclusive_Scan_Sum(Num_ContactD, number_of_contacts_possible);
	potentialCollisions.resize(number_of_contacts_possible);
	cout << "Number of possible collisions: " << number_of_contacts_possible << endl;
	// END STEP 5

	// STEP 6: Store the possible AABB collision pairs
#ifdef SIM_ENABLE_GPU_MODE
	device_Store_AABB_AABB_Intersection __KERNEL__(BLOCKS(last_active_bin),THREADS)(
			CASTF3(aabb_data),
			CASTU1(bin_number),
			CASTU1(body_number),
			CASTU1(bin_start_index),
			CASTU1(Num_ContactD),
			CASTLL(potentialCollisions));
#else
	host_Store_AABB_AABB_Intersection(aabb_data.data(),
			bin_number.data(),
			body_number.data(),
			bin_start_index.data(),
			Num_ContactD.data(),
			potentialCollisions.data());
#endif
	thrust::sort(potentialCollisions.begin(), potentialCollisions.end());
	number_of_contacts_possible = thrust::unique(potentialCollisions.begin(),
			potentialCollisions.end()) - potentialCollisions.begin();
	cout << "Number of possible collisions: " << number_of_contacts_possible << endl;
	// END STEP 6

	double endTime = omp_get_wtime();
	printf("Time to detect: %lf seconds\n", (endTime - startTime));
	return 0;
}
//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
